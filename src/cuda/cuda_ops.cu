#include "hip/hip_runtime.h"
#include "cuda_ops.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <stdio.h>

// Check if CUDA is available
int cuda_available() {
    int count;
    hipError_t err = hipGetDeviceCount(&count);
    return (err == hipSuccess && count > 0);
}

// CUDA error checking macro
#define CHECK_CUDA(err) do { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__, hipGetErrorString(err_)); \
        exit(1); \
    } \
} while (0)

// CUDA kernel for element-wise addition
__global__ void cuda_matrix_add_kernel(float* a, const float* b, size_t rows, size_t cols, size_t stride_a, size_t stride_b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < rows && j < cols) {
        a[i * stride_a + j] += b[i * stride_b + j];
    }
}

// CUDA kernel for ReLU activation
__global__ void cuda_matrix_relu_kernel(float* m, size_t rows, size_t cols, size_t stride) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < rows && j < cols) {
        float val = m[i * stride + j];
        m[i * stride + j] = val > 0 ? val : 0;
    }
}

// CUDA kernel for sigmoid activation
__global__ void cuda_matrix_sigmoid_kernel(float* m, size_t rows, size_t cols, size_t stride) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < rows && j < cols) {
        m[i * stride + j] = 1.0f / (1.0f + expf(-m[i * stride + j]));
    }
}

// CUDA kernel for softmax activation (per row)
__global__ void cuda_matrix_softmax_kernel(float* m, size_t rows, size_t cols, size_t stride) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < rows) {
        // Find max value in row for numerical stability
        float max_val = m[i * stride];
        for (int j = 1; j < cols; j++) {
            if (m[i * stride + j] > max_val) {
                max_val = m[i * stride + j];
            }
        }
        
        // Compute exponentials and sum
        float sum = 0.0f;
        for (int j = 0; j < cols; j++) {
            float val = expf(m[i * stride + j] - max_val);
            m[i * stride + j] = val;
            sum += val;
        }
        
        // Normalize
        for (int j = 0; j < cols; j++) {
            m[i * stride + j] /= sum;
        }
    }
}

// Allocate matrix on GPU
void cuda_matrix_alloc(Matrix* m) {
    CHECK_CUDA(hipMalloc(&m->data, m->rows * m->cols * sizeof(float)));
}

// Free matrix from GPU
void cuda_matrix_free(Matrix* m) {
    CHECK_CUDA(hipFree(m->data));
}

// Copy matrix from host to device
void cuda_matrix_copy_to_gpu(const Matrix* host_src, Matrix* device_dst) {
    CHECK_CUDA(hipMemcpy(device_dst->data, host_src->data, 
                         host_src->rows * host_src->cols * sizeof(float),
                         hipMemcpyHostToDevice));
}

// Copy matrix from device to host
void cuda_matrix_copy_to_cpu(const Matrix* device_src, Matrix* host_dst) {
    CHECK_CUDA(hipMemcpy(host_dst->data, device_src->data,
                         device_src->rows * device_src->cols * sizeof(float),
                         hipMemcpyDeviceToHost));
}

// Matrix addition on GPU
void cuda_matrix_add(Matrix* a, const Matrix* b) {
    dim3 blockSize(16, 16);
    dim3 gridSize((a->rows + blockSize.x - 1) / blockSize.x,
                  (a->cols + blockSize.y - 1) / blockSize.y);
    
    cuda_matrix_add_kernel<<<gridSize, blockSize>>>(
        a->data, b->data, a->rows, a->cols, a->stride, b->stride
    );
    CHECK_CUDA(hipGetLastError());
}

// Matrix multiplication using cuBLAS
void cuda_matrix_multiply(const Matrix* a, const Matrix* b, Matrix* c) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    float alpha = 1.0f;
    float beta = 0.0f;
    
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                c->cols, c->rows, a->cols,
                &alpha,
                b->data, b->stride,
                a->data, a->stride,
                &beta,
                c->data, c->stride);
    
    hipblasDestroy(handle);
}

// ReLU activation on GPU
void cuda_matrix_relu(Matrix* m) {
    dim3 blockSize(16, 16);
    dim3 gridSize((m->rows + blockSize.x - 1) / blockSize.x,
                  (m->cols + blockSize.y - 1) / blockSize.y);
    
    cuda_matrix_relu_kernel<<<gridSize, blockSize>>>(
        m->data, m->rows, m->cols, m->stride
    );
    CHECK_CUDA(hipGetLastError());
}

// Sigmoid activation on GPU
void cuda_matrix_sigmoid(Matrix* m) {
    dim3 blockSize(16, 16);
    dim3 gridSize((m->rows + blockSize.x - 1) / blockSize.x,
                  (m->cols + blockSize.y - 1) / blockSize.y);
    
    cuda_matrix_sigmoid_kernel<<<gridSize, blockSize>>>(
        m->data, m->rows, m->cols, m->stride
    );
    CHECK_CUDA(hipGetLastError());
}

// Softmax activation on GPU
void cuda_matrix_softmax(Matrix* m) {
    dim3 blockSize(256);
    dim3 gridSize((m->rows + blockSize.x - 1) / blockSize.x);
    
    cuda_matrix_softmax_kernel<<<gridSize, blockSize>>>(
        m->data, m->rows, m->cols, m->stride
    );
    CHECK_CUDA(hipGetLastError());
}